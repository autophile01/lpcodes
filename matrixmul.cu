#include "hip/hip_runtime.h"
!nvcc --version

!pip install git+https://github.com/andreinechaev/nvcc4jupyter.git

%load_ext nvcc4jupyter
%%cuda
#include <iostream>  // for input/output operations, 
#include <vector>    // for using vectors (arrays, lists etc)
#include <hip/hip_runtime.h> // provides CUDA runtime APIs for CUDA programming
using namespace std;

// Kernel function for matrix multiplication
__global__ void matrixMul(const int* A, const int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];  //IMPORTANT TO REMEMBER
        }
        C[row * N + col] = sum;  //IMPORTANT TO REMEMBER
    }
}

// Function to print a matrix
void printMatrix(const std::vector<int>& matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << matrix[i * N + j] << " ";  //IMPORTANT TO REMEMBER
        }
        std::cout << std::endl;
    }
}

int main() {
    const int N = 3; // Size of the matrices (for demonstration purposes)
    const int blockSize = 2; // Threads per block
    const int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks

    // Initialize host matrices
    std::vector<int> hostA = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    std::vector<int> hostB = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    std::vector<int> hostC(N * N);

    // Print input matrices
    std::cout << "Matrix A:" << std::endl;
    printMatrix(hostA, N);
    std::cout << "Matrix B:" << std::endl;
    printMatrix(hostB, N);

    // Declare device pointers
    int *deviceA, *deviceB, *deviceC;

    // Allocate device memory
    hipMalloc((void**)&deviceA, N * N * sizeof(int));
    hipMalloc((void**)&deviceB, N * N * sizeof(int));
    hipMalloc((void**)&deviceC, N * N * sizeof(int));

    // Copy host data to device
    hipMemcpy(deviceA, hostA.data(), N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB.data(), N * N * sizeof(int), hipMemcpyHostToDevice);

    //IMPORTANT TO REMEMBER
    // Launch kernel                                  
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 numBlocks(gridSize, gridSize);
    matrixMul<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, deviceC, N);

    // Copy result back to host
    hipMemcpy(hostC.data(), deviceC, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result matrix
    std::cout << "Matrix C (A + B):" << std::endl;
    printMatrix(hostC, N);

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}